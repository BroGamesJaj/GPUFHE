
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(long long int* a,long long int* b,long long int* c){
    int i = threadIdx.x + blockIdx.y * blockDim.x;
    c[i] = a[i] + b[i];
}

__managed__ long long int vector_a[1073741824], vector_b[1073741824], vector_c[1073741824];

int main(){
    for (int i = 0; i < 1073741824; i++){
        vector_a[i] = i;
        vector_b[i] = 1073741824 - i;
    }

    add<<<1,256>>>(vector_a, vector_b, vector_c);

    hipDeviceSynchronize();

    
    long long int res = 0;
    for (int i = 0; i < 1073741824; i++){
        res += vector_c[i];
    }
    printf("sum result: % d",res);
}