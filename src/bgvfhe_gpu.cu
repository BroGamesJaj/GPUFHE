
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int* a, int* b, int* c){
    int i = threadIdx.x + blockIdx.y * blockDim.x;
    c[i] = a[i] + b[i];
}

__managed__ int vector_a[4294967296], vector_b[4294967296], vector_c[4294967296];

int main(){
    for (int i = 0; i < 4294967296; i++){
        vector_a[i] = i;
        vector_b[i] = 4294967296 - i;
    }

    add<<<1,256>>>(vector_a, vector_b, vector_c);

    hipDeviceSynchronize();

    
    int res = 0;
    for (int i = 0; i < 4294967296; i++){
        res += vector_c[i];
    }
    printf("sum result: % d",res);
}