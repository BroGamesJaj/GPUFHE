#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include "sub/poly.h"
#include "sub/poly_eqs.h"
#include <random>
#include <inttypes.h>
#include <hip/hip_runtime.h>
#define N 10000

void init_poly(int64_t *array, int n) {
    std::random_device rd;                     // Seed for randomness
    std::mt19937 gen(rd());                    // Mersenne Twister generator
    std::uniform_int_distribution<size_t> dis(1, 10);
    for (size_t i = 0; i < n; ++i) {
        array[i] = dis(gen); // Generate random number and assign to array
    }
}

double get_time() {
    auto now = std::chrono::high_resolution_clock::now();
    auto duration = now.time_since_epoch();
    return std::chrono::duration<double>(duration).count();
}
void AddTest(){
    printf("test for polynomial addition\n");
    size_t size1 = N * sizeof(int64_t);
    size_t size_out = N * sizeof(int64_t);
    Polinomial array(N);
    Polinomial array2(N);
    Polinomial array3(N);
    Polinomial array_gpu(N);
    int64_t *d_a, *d_b, *d_c;

    printf("Benchmarking CPU implementation...\n");
    double cpu_total_time = 0.0;
    for (int i = 0; i < 20; i++) {
        double start_time = get_time();
        array3 = poly_eqs::PolyAdd_cpu(array,array2);
        double end_time = get_time();
        cpu_total_time += end_time - start_time;
    }
    double cpu_avg_time = cpu_total_time / 20.0;
    
    printf("\n");
    hipMalloc(&d_a, size1);
    hipMalloc(&d_b, size1);
    hipMalloc(&d_c, size_out);
    hipMemset(d_c, 0, size_out);
    hipMemcpy(d_a, array.getCoeffPointer(), size1, hipMemcpyHostToDevice );
    hipMemcpy(d_b, array2.getCoeffPointer(), size1, hipMemcpyHostToDevice );

    int block_num = (N + 256 - 1) / 256;

    printf("Benchmarking GPU implementation...\n");
    double gpu_total_time = 0.0;
    for (int i = 0; i < 20; i++) {
        double start_time = get_time();
        poly_eqs::PolyAdd_gpu<<<block_num,256>>>(d_a, d_b, d_c);
        hipDeviceSynchronize();
        double end_time = get_time();
        gpu_total_time += end_time - start_time;
    }
    double gpu_avg_time = gpu_total_time / 20.0;
    hipMemcpy(array_gpu.getCoeffPointer(), d_c, size_out, hipMemcpyDeviceToHost);

    bool correct = true;
    for (int i = 0; i < array_gpu.getSize(); i++) {
        if(array_gpu[i] - array3[i] != 0){
            correct = false;
            break;
        }
    }
    printf("CPU average time: %f milliseconds\n", cpu_avg_time*1000);
    printf("GPU average time: %f milliseconds\n", gpu_avg_time*1000);
    printf("Speedup: %fx\n", cpu_avg_time / gpu_avg_time);
    printf("Results are %s\n", correct ? "correct" : "incorrect");
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    printf("\n");
}


void SubTest(){
    printf("Test for Polynomial substration\n");
    size_t size1 = N * sizeof(int64_t);
    size_t size_out = N * sizeof(int64_t);
    Polinomial array(N);
    Polinomial array2(N);
    Polinomial array3(N);
    Polinomial array_gpu(N);
    int64_t *d_a, *d_b, *d_c;

    printf("Benchmarking CPU implementation...\n\n");
    double cpu_total_time = 0.0;
    for (int i = 0; i < 20; i++) {
        double start_time = get_time();
        array3 = poly_eqs::PolyAdd_cpu(array,array2);
        double end_time = get_time();
        cpu_total_time += end_time - start_time;
    }
    double cpu_avg_time = cpu_total_time / 20.0;

    hipMalloc(&d_a, size1);
    hipMalloc(&d_b, size1);
    hipMalloc(&d_c, size_out);
    hipMemset(d_c, 0, size_out);
    hipMemcpy(d_a, array.getCoeffPointer(), size1, hipMemcpyHostToDevice );
    hipMemcpy(d_b, array2.getCoeffPointer(), size1, hipMemcpyHostToDevice );

    int block_num = (N + 256 - 1) / 256;

    printf("Benchmarking GPU implementation...\n");
    double gpu_total_time = 0.0;
    for (int i = 0; i < 20; i++) {
        double start_time = get_time();
        poly_eqs::PolyAdd_gpu<<<block_num,256>>>(d_a, d_b, d_c);
        hipDeviceSynchronize();
        double end_time = get_time();
        gpu_total_time += end_time - start_time;
    }
    double gpu_avg_time = gpu_total_time / 20.0;
    hipMemcpy(array_gpu.getCoeffPointer(), d_c, size_out, hipMemcpyDeviceToHost);

    bool correct = true;
    for (int i = 0; i < array_gpu.getSize(); i++) {
        if(array_gpu[i] - array3[i] != 0){
            correct = false;
            break;
        }
    }
    printf("CPU average time: %f milliseconds\n", cpu_avg_time*1000);
    printf("GPU average time: %f milliseconds\n", gpu_avg_time*1000);
    printf("Speedup: %fx\n", cpu_avg_time / gpu_avg_time);
    printf("Results are %s\n", correct ? "correct" : "incorrect");
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    printf("\n");
}
 
void MultTest(){
    printf("test for polynomial multiplication\n");
    size_t size1 = N * sizeof(int64_t);
    size_t size_out = (2 * N - 1) * sizeof(int64_t);
    Polinomial array(N);
    Polinomial array2(N);
    Polinomial array3((2 * N -1));
    Polinomial array_gpu((2 * N -1));
    int64_t *d_a, *d_b, *d_c;


    init_poly(array.getCoeffPointer(), array.getSize()); 
    init_poly(array2.getCoeffPointer(), array2.getSize()); 

    printf("Benchmarking CPU implementation...\n");
    double cpu_total_time = 0.0;
    for (int i = 0; i < 20; i++) {
        double start_time = get_time();
        array3 = poly_eqs::PolyMult_cpu(array,array2);
        double end_time = get_time();
        cpu_total_time += end_time - start_time;
    }
    double cpu_avg_time = cpu_total_time / 20.0;
    
    printf("\n");
    hipMalloc(&d_a, size1);
    hipMalloc(&d_b, size1);
    hipMalloc(&d_c, size_out);
    hipMemset(d_c, 0, size_out);
    hipMemcpy(d_a, array.getCoeffPointer(), size1, hipMemcpyHostToDevice );
    hipMemcpy(d_b, array2.getCoeffPointer(), size1, hipMemcpyHostToDevice );

    int block_num = (2 * N + 256 - 1) / 256;

    printf("Benchmarking GPU implementation...\n");
    double gpu_total_time = 0.0;
    for (int i = 0; i < 20; i++) {
        double start_time = get_time();
        poly_eqs::PolyMult_gpu<<<block_num,256>>>(d_a, d_b, d_c, array.getSize());
        hipDeviceSynchronize();
        double end_time = get_time();
        gpu_total_time += end_time - start_time;
    }
    double gpu_avg_time = gpu_total_time / 20.0;
    hipMemcpy(array_gpu.getCoeffPointer(), d_c, size_out, hipMemcpyDeviceToHost);

    bool correct = true;
    for (int i = 0; i < array_gpu.getSize(); i++) {
        if(array_gpu[i] - array3[i] != 0){
            correct = false;
            break;
        }
    }
    printf("CPU average time: %f milliseconds\n", cpu_avg_time*1000);
    printf("GPU average time: %f milliseconds\n", gpu_avg_time*1000);
    printf("Speedup: %fx\n", cpu_avg_time / gpu_avg_time);
    printf("Results are %s\n", correct ? "correct" : "incorrect");
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    printf("\n");
}

void DivTest() {
    printf("Test for Polynomial substration\n");
    Polinomial dividend(N);
    Polinomial divisor(N);


    init_poly(dividend.getCoeffPointer(), dividend.getSize());
    init_poly(divisor.getCoeffPointer(), divisor.getSize());

    std::pair<Polinomial, Polinomial> res = poly_eqs::PolyDiv_cpu(dividend, divisor);

    printf("Benchmarking CPU implementation...\n");
    double cpu_total_time = 0.0;
    for (int i = 0; i < 20; i++) {
        double start_time = get_time();
        res = poly_eqs::PolyDiv_cpu(dividend, divisor);
        double end_time = get_time();
        cpu_total_time += end_time - start_time;
    }
    double cpu_avg_time = cpu_total_time / 20.0;

    std::pair<Polinomial, Polinomial> res_gpu = poly_eqs::PolyDiv_gpu(dividend, divisor);

    printf("Benchmarking GPU implementation...\n");
    double gpu_total_time = 0.0;
    for (int i = 0; i < 20; i++) {
        double start_time = get_time();
        res_gpu = poly_eqs::PolyDiv_gpu(dividend, divisor);
        double end_time = get_time();
        gpu_total_time += end_time - start_time;
    }
    double gpu_avg_time = gpu_total_time / 20.0;

    printf("Benchmarking Device Pointer GPU implementation...\n");
    int64_t *remainder_d, *divisor_d;
    int64_t* quotient = (int64_t*)malloc(sizeof(int64_t) * dividend.getSize() - divisor.getSize() + 1);
    hipMalloc(&remainder_d, sizeof(int64_t) * dividend.getSize());
    hipMalloc(&divisor_d, sizeof(int64_t) * divisor.getSize());
    hipMemcpy(remainder_d, dividend.getCoeffPointer(), sizeof(int64_t) * dividend.getSize(), hipMemcpyHostToDevice);
    hipMemcpy(divisor_d, divisor.getCoeffPointer(), sizeof(int64_t) * divisor.getSize(), hipMemcpyHostToDevice);
    poly_eqs::PolyDiv_gpu(remainder_d, quotient, divisor_d, dividend.getSize(), divisor.getSize());
    double gpu2_total_time = 0.0;
    for (int i = 0; i < 20; i++) {
        hipMemcpy(remainder_d, dividend.getCoeffPointer(), sizeof(int64_t) * dividend.getSize(), hipMemcpyHostToDevice);
        hipMemcpy(divisor_d, divisor.getCoeffPointer(), sizeof(int64_t) * divisor.getSize(), hipMemcpyHostToDevice);
        double start_time = get_time();
        poly_eqs::PolyDiv_gpu(remainder_d, quotient, divisor_d, dividend.getSize(), divisor.getSize());
        double end_time = get_time();
        gpu2_total_time += end_time - start_time;
    }
    double gpu2_avg_time = gpu2_total_time / 20.0;

    bool correct = true;
    for (int i = 0; i < res_gpu.first.getSize(); i++) {
        if(res_gpu.first[i] - res.first[i] != 0){
            correct = false;
            break;
        }
    }

    bool correct2 = true;
    for (int i = 0; i < dividend.getSize() - divisor.getSize() + 1; i++) {
        if(quotient[i] - res.first[i] != 0){
            correct2 = false;
            break;
        }
    }
    printf("CPU average time: %f milliseconds\n", cpu_avg_time*1000);
    printf("GPU average time: %f milliseconds\n", gpu_avg_time*1000);
    printf("GPU Device Pointer average time: %f milliseconds\n", gpu2_avg_time*1000);
    printf("Speedup: %fx\n", cpu_avg_time / gpu2_avg_time);
    printf("Results are %s\n", correct ? "correct" : "incorrect");
    printf("Results2 are %s\n", correct2 ? "correct" : "incorrect");
}

int main(){
    DivTest();
    //SubTest();
    //AddTest();
    //MultTest();
    return 0;
}