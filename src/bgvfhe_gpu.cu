#include "hip/hip_runtime.h"
#include <stdio.h>
#include "sub/poly.h"
#include "sub/poly_eqs.h"
#include <random>
#include <inttypes.h>
#define N 1000

__global__ void add(int* a, int* b, int* c){
    int i = threadIdx.x + blockIdx.y * blockDim.x;
    c[i] = a[i] + b[i];
}

__global__ void PolyMult_gpu(uint64_t* a, uint64_t* b, uint64_t* c, int size){
    int i = threadIdx.x + blockIdx.y * blockDim.x;
    if(i < size){
        c[i]  = a[i] * b[i];
    }
}
void init_poly(uint64_t *array, int n) {
    std::random_device rd;                     // Seed for randomness
    std::mt19937 gen(rd());                    // Mersenne Twister generator
    std::uniform_int_distribution<size_t> dis(1, 10);
    for (size_t i = 0; i < n; ++i) {
        array[i] = dis(gen); // Generate random number and assign to array
    }
}
int main(){
    size_t size = N * sizeof(uint64_t);
    Polinomial array(size);
    Polinomial array2(size);
    Polinomial array3(size+size-1);
    uint64_t *d_a, *d_b, *d_c;
    std::random_device rd;                     // Seed for randomness
    std::mt19937 gen(rd());                    // Mersenne Twister generator
    std::uniform_int_distribution<size_t> dis(1, 10); // Uniform distribution [1, 10]

    init_poly(array.getCoeffPointer(), array.getSize()); 
    init_poly(array2.getCoeffPointer(), array2.getSize()); 
    array3 = poly_eqs::PolyMult(array,array2);
    for (int i=0; i<array3.getSize(); i++) 
    { 
       printf( "%" PRIu64, array3[i]); 
       if (i != 0) 
        printf("x^%d",i) ; 
       if (i != array3.getSize()-1) 
       printf(" + "); 
    } 

    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size+size-1);
    
    hipMemcpy(d_a, array.getCoeffPointer(), size, hipMemcpyHostToDevice );
    hipMemcpy(d_b, array2.getCoeffPointer(), size, hipMemcpyHostToDevice );

    int block_num = (N + 256 - 1) / 256;
    PolyMult_gpu<<<block_num,256>>>(d_a, d_b, d_c, N);
    hipDeviceSynchronize();

    hipMemcpy(array3.getCoeffPointer(), d_c, size+size-1, hipMemcpyDeviceToHost);

    for (int i=0; i<array3.getSize(); i++) 
    { 
       printf( "%" PRIu64, array3[i]); 
       if (i != 0) 
        printf("x^%d",i) ; 
       if (i != array3.getSize()-1) 
       printf(" + "); 
    } 
}