
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int* a, int* b, int* c){
    int i = threadIdx.x + blockIdx.y * blockDim.x;
    c[i] = a[i] + b[i];
}

__managed__ int vector_a[256], vector_b[256], vector_c[256];

int main(){
    for (int i = 0; i < 256; i++){
        vector_a[i] = i;
        vector_b[i] = 256 - i;
    }

    add<<<1,256>>>(vector_a, vector_b, vector_c);

    hipDeviceSynchronize();

    
    int res = 0;
    for (int i = 0; i < 256; i++){
        printf("vector_c %d %d",i,vector_c[i]);
        res += vector_c[i];
    }
    printf("sum result: % d",res);
}