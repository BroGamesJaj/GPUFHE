#include "hip/hip_runtime.h"
#include "bgvfhe_gpu.cuh"
#define N 10



double get_time() {
    auto now = std::chrono::high_resolution_clock::now();
    auto duration = now.time_since_epoch();
    return std::chrono::duration<double>(duration).count();
}

bool computeNoiseNorm(const Polinomial& poly) {

    int size = poly.getSize();  // Get the size of the polynomial

    int max_noise = 0;
    for (int i = 0; i < size; ++i) {
        if(poly[i] > max_noise) max_noise = poly[i];
    }
    if(max_noise < poly.getCoeffModulus() / 2){
        printf("noise okay\n");
    }else{
        printf("noise bad\n");
    }
    return max_noise < poly.getCoeffModulus() / 2;  // L2 norm (Euclidean norm)
}


Polinomial GeneratePrivateKey(int64_t coeff_modulus, GeneralArray<int64_t> poly_modulus){
    if(coeff_modulus != 0 && poly_modulus.getSize() != 0){
        Polinomial randomPoly = poly::randomTernaryPoly(coeff_modulus, poly_modulus);
        return randomPoly;
    }else{
        throw std::runtime_error("coefficient or poly_modulus is not set");
    }
}

std::pair<Polinomial,Polinomial> GeneratePublicKey(Polinomial& sk, int64_t coeff_modulus, GeneralArray<int64_t>& poly_modulus, int64_t plaintext_modulus){
    Polinomial e = poly::randomNormalPoly(coeff_modulus,poly_modulus);
    //printf("e noise: ");
    //computeNoiseNorm(e);
    Polinomial a = poly::randomTernaryPoly(coeff_modulus,poly_modulus);

    Polinomial temp1 = poly_eqs::PolyMult_cpu(a,sk);
    //printf("temp1 noise: ");
    //computeNoiseNorm(temp1);

    Polinomial temp2 = poly_eqs::PolyMult_cpu(e,plaintext_modulus);
    //printf("temp2 noise: ");
    //computeNoiseNorm(temp2);

    Polinomial b = poly_eqs::PolyAdd_cpu(temp1,temp2);
    //computeNoiseNorm(b);
    b.modCenter();

    //PublicKeyTest(b,a,sk,a,e,plaintext_modulus);
    return std::make_pair(b,-a);
}



bool isSmallNorm(const Polinomial& poly, int64_t bound) {
    for (int64_t coef : poly.getCoeff()) { // Iterate over coefficients of the polynomial
        if (std::abs(coef) > bound) {
            return false; // Coefficient exceeds the allowed bound
        }
    }
    return true; // All coefficients are within the bound
}

std::pair<Polinomial, Polinomial> asymetricEncryption(Polinomial pk0, Polinomial pk1, Polinomial msg, int64_t plaintext_modulus, int64_t coef_modulus, GeneralArray<int64_t> poly_modulus, int64_t degree){
    Polinomial u = poly::randomTernaryPoly(coef_modulus,poly_modulus);
    Polinomial e0 = poly::randomNormalPoly(coef_modulus,poly_modulus,coef_modulus/static_cast<int>(pow(2, degree)));
    Polinomial e1 = poly::randomNormalPoly(coef_modulus,poly_modulus);
    Polinomial c0_temp1 = poly_eqs::PolyMult_cpu(pk0,u);

    Polinomial c0_temp2 = poly_eqs::PolyMult_cpu(e0,plaintext_modulus);

    Polinomial c0 = poly_eqs::PolyAdd_cpu(poly_eqs::PolyAdd_cpu(c0_temp1,c0_temp2),msg);
    //printf("c0 noise: ");
    //computeNoiseNorm(c0);

    Polinomial c1_temp1 = poly_eqs::PolyMult_cpu(pk1,u);

    Polinomial c1_temp2 = poly_eqs::PolyMult_cpu(e1,plaintext_modulus);

    Polinomial c1 = poly_eqs::PolyAdd_cpu(c1_temp1,c1_temp2);
    //printf("c1 noise: ");
    //computeNoiseNorm(c1);

    //printf("c0\n");
    //c0.print();
    //c1.print();
    return std::make_pair(c0,c1);
}

Polinomial decrypt(Polinomial c0, Polinomial c1, Polinomial sk, int64_t plaintext_modulus){
    Polinomial sk_c1 = poly_eqs::PolyMult_cpu(c1,sk);
    //printf("sk_c1 noise: ");
    //computeNoiseNorm(sk_c1);
    Polinomial msg = poly_eqs::PolyAdd_cpu(c0,sk_c1);
    //computeNoiseNorm(msg);

    msg.modCenter(plaintext_modulus);
    
    return msg;
}


//decrypting multiplied msgs
Polinomial decrypt_quad(Polinomial c0, Polinomial c1, Polinomial c2, Polinomial sk, int64_t plaintext_modulus){
    Polinomial sk_c1 = poly_eqs::PolyMult_cpu(c1,sk);
    Polinomial sk_c2 = poly_eqs::PolyMult_cpu(c2,sk);
     Polinomial sk_sk_c1 = poly_eqs::PolyMult_cpu(sk_c2,sk);
    //printf("sk_c1 noise: ");
    //computeNoiseNorm(sk_c1);
    Polinomial msg = poly_eqs::PolyAdd_cpu(poly_eqs::PolyAdd_cpu(c0,sk_c1),sk_sk_c1);
    //computeNoiseNorm(msg);

    msg.modCenter(plaintext_modulus);    
    return msg;
}

int64_t logBase(int64_t value, int base) {
    if (value <= 0) {
        throw std::invalid_argument("Value must be positive.");
    }
    if (base <= 1) {
        throw std::invalid_argument("Base must be greater than 1.");
    }

    int64_t result = 0;
    while (value >= base) {
        value /= base;
        ++result;
    }

    return result;
}

GeneralArray<int64_t> int2Base(int value, int base, int& digitCount) {
    // Calculate number of digits required
    digitCount = 0;
    int temp = value;
    while (temp > 0) {
        temp /= base;
        ++digitCount;
    }
    if (digitCount == 0) digitCount = 1; // Handle the case for value = 0

    // Allocate memory for the digits
    GeneralArray<int64_t> digits(digitCount);
    temp = value;

    // Extract digits
    for (int i = 0; i < digitCount; ++i) {
        digits[i] = temp % base;
        temp /= base;
    }

    return digits;
}

GeneralArray<Polinomial*> poly2Base(Polinomial poly, int base){
    int n_terms = ceil(logBase(poly.getCoeffModulus(),base));
    int degree = poly.getPolyModSize() - 1;
    if(degree <= 0 && n_terms <= 0){
        printf("Poly2Base: degree or n_terms cannot be 0");
    }
    GeneralArray<GeneralArray<int64_t>> coeffs(degree);

    for (int i = 0; i < degree; ++i) {
        coeffs[i] = GeneralArray<int64_t>(n_terms);
        for (int j = 0; j < n_terms; j++) {
            coeffs[i][j] = 0;
        }
    }


    for (int i = 0; i < degree; ++i){
        int digitCount = 0;
        GeneralArray<int64_t> digits = int2Base(poly[i] % poly.getCoeffModulus(),base,digitCount);
        for (int j = 0; j < digitCount; ++j) {
            coeffs[i][j] = digits[j];
        }
        for (int j = digitCount; j < n_terms; ++j) {
            coeffs[i][j] = 0;
        }
    }

    GeneralArray<Polinomial*> poly_list(n_terms);
    for (size_t i = 0; i < n_terms; i++) {
        poly_list[i] = new Polinomial(degree,poly.getCoeffModulus(),poly.getPolyModulus());
        for (size_t j = 0; j < degree; j++) {
            (*poly_list[i])[j] = coeffs[j][i];
        }
    }

    return poly_list;
}


std::pair<Polinomial,Polinomial> Relinearization(Polinomial c0, Polinomial c1, Polinomial c2, GeneralArray<std::pair<Polinomial,Polinomial>*> eks, int base, int64_t coef_modulus, int64_t poly_modulus){
    auto c2_polys = poly2Base(c2,base);

    Polinomial c0_hat = c0;
    Polinomial c1_hat = c1;
    for (size_t i = 0; i < eks.getSize(); i++) {
        c0_hat = poly_eqs::PolyAdd_cpu(c0_hat, poly_eqs::PolyMult_cpu(*c2_polys[i],(*eks[i]).first));
        c1_hat = poly_eqs::PolyAdd_cpu(c1_hat, poly_eqs::PolyMult_cpu(*c2_polys[i],(*eks[i]).second));
    }
    return std::make_pair(c0_hat,c1_hat);
}


bool isNoiseSmallEnough(const Polinomial& noise, double threshold) {
    double norm = computeNoiseNorm(noise);
    return norm < threshold;  // Check if the noise norm is below the threshold
}

int main(){

    double start_time = get_time();
    //cleartext_encoding::ClearTextEncodingTest();
    printf("started\n");
    int64_t n = 2048; // degree of the polynomials
    int64_t coef_modulus = pow(2,40); // can the second value if you want to change the size of q(coefficient_modulus)
    int64_t plaintext_modulus = pow(2,30); // max size of stored values (max 32 if no operations on poly)
    int64_t max_degree = 16; // amount of numbers stored
    int base = 12;

    for (size_t i = 0; i < 5; i++) {}
    GeneralArray<int64_t> poly_modulus = poly::initPolyModulus(n); 

    
    Polinomial sk = GeneratePrivateKey(coef_modulus, poly_modulus);
    auto pk = GeneratePublicKey(sk, coef_modulus, poly_modulus, plaintext_modulus);
    printf("PK generator ended\n");

    Polinomial msg = poly::randomUniformPolyMSG(coef_modulus, poly_modulus, plaintext_modulus/100000,max_degree > n ? n : max_degree);
    Polinomial msg2 = poly::randomUniformPolyMSG(coef_modulus, poly_modulus, plaintext_modulus/100000,max_degree > n ? n : max_degree);
    printf("MSG:\n");
    msg.print();
    printf("MSG2:\n");
    msg2.print();
    auto e_msg = asymetricEncryption(pk.first,pk.second,msg,plaintext_modulus,coef_modulus,poly_modulus,n);
    auto e_msg2 = asymetricEncryption(pk.first,pk.second,msg2,plaintext_modulus,coef_modulus,poly_modulus,n);
    auto mult_res = cMult_cpu(e_msg, e_msg2);
    Polinomial d_msg = decrypt(e_msg.first, e_msg.second,sk,plaintext_modulus);
    printf("decrypted MSG1:\n");
    d_msg.print();
    Polinomial d_msg2 = decrypt(e_msg2.first, e_msg2.second,sk,plaintext_modulus);
    printf("decrypted MSG2:\n");
    d_msg2.print();


    printf("Benchmarking CPU implementation...\n");
        double cpu_total_time = 0.0;
        for (int i = 0; i < 20; i++) {
            double start_time2 = get_time();
            double end_time2 = get_time();
            cpu_total_time += end_time2 - start_time2;
        }
        double cpu_avg_time = cpu_total_time / 20.0;
    double end_time = get_time();
    printf("CPU average time: %f milliseconds\n", cpu_avg_time*1000);
    printf("CPU run time: %f milliseconds\n", (end_time - start_time)*1000);

}