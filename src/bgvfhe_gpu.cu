#include "hip/hip_runtime.h"
#include <stdio.h>
#include "sub/poly.h"
#include "sub/poly_eqs.h"
#include <random>
#include <inttypes.h>
#define N 1000000
#define M 1

__global__ void add(int* a, int* b, int* c){
    int i = threadIdx.x + blockIdx.y * blockDim.x;
    c[i] = a[i] + b[i];
}

__global__ void PolyMult_gpu(uint64_t* poly_1, uint64_t* poly_2, uint64_t* result, size_t poly_1_size, size_t poly_2_size){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= poly_1_size + poly_2_size - 1) return;
    uint64_t sum = 0;
    for (int j = 0; j < poly_1_size; j++) {
        if (i - j >= 0 && i - j < poly_2_size) {
            sum += poly_1[j] * poly_2[i - j];
        }
    }
    result[i] = sum;
}
void init_poly(uint64_t *array, int n) {
    std::random_device rd;                     // Seed for randomness
    std::mt19937 gen(rd());                    // Mersenne Twister generator
    std::uniform_int_distribution<size_t> dis(1, 10);
    for (size_t i = 0; i < n; ++i) {
        array[i] = dis(gen); // Generate random number and assign to array
    }
}
int main(){
    size_t size1 = N * sizeof(uint64_t);
    size_t size2 = M * sizeof(uint64_t);
    size_t size_out = (M + N - 1) * sizeof(uint64_t);
    Polinomial array(N);
    Polinomial array2(M);
    Polinomial array3((M + N -1));
    uint64_t *d_a, *d_b, *d_c;

    init_poly(array.getCoeffPointer(), array.getSize()); 
    init_poly(array2.getCoeffPointer(), array2.getSize()); 
    for (int i=0; i<array.getSize(); i++) 
    { 
       printf( "%" PRIu64, array[i]); 
       if (i != 0) 
        printf("x^%d",i) ; 
       if (i != array.getSize()-1) 
       printf(" + "); 
    } 
    printf("\n");
    for (int i=0; i<array2.getSize(); i++) 
    { 
       printf( "%" PRIu64, array2[i]); 
       if (i != 0) 
        printf("x^%d",i) ; 
       if (i != array2.getSize()-1) 
       printf(" + "); 
    } 
    printf("\n");
    printf("\n");
    array3 = poly_eqs::PolyMult(array,array2);
    for (int i=0; i<array3.getSize(); i++) 
    { 
       printf( "%" PRIu64, array3[i]); 
       if (i != 0) 
        printf("x^%d",i) ; 
       if (i != array3.getSize()-1) 
       printf(" + "); 
    } 
    printf("\n");
    printf("\n");
    hipMalloc(&d_a, size1);
    hipMalloc(&d_b, size2);
    hipMalloc(&d_c, size_out);
    hipMemset(d_c, 0, size_out);
    hipMemcpy(d_a, array.getCoeffPointer(), size1, hipMemcpyHostToDevice );
    hipMemcpy(d_b, array2.getCoeffPointer(), size2, hipMemcpyHostToDevice );

    int block_num = (M * N + 256 - 1) / 256;
    PolyMult_gpu<<<block_num,256>>>(d_a, d_b, d_c, array.getSize(), array.getSize());
    hipDeviceSynchronize();

    hipMemcpy(array3.getCoeffPointer(), d_c, size_out, hipMemcpyDeviceToHost);

    for (int i=0; i<array3.getSize(); i++) 
    { 
       printf( "%" PRIu64, array3[i]); 
       if (i != 0) 
        printf("x^%d",i) ; 
       if (i != array3.getSize()-1) 
       printf(" + "); 
    } 

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}