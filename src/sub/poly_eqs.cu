#include "hip/hip_runtime.h"
#include "poly_eqs.h"

namespace poly_eqs{
    Polinomial PolyMult_cpu(Polinomial p1, Polinomial p2){
        Polinomial prod(p1.getSize()+p2.getSize()-1);

        for (int i=0; i<p1.getSize(); i++) { 
            for (int j=0; j<p2.getSize(); j++){
                prod[i+j] += p1[i]*p2[j]; 
            }
        } 
        return prod;
    }

    __global__ void PolyMult_gpu(uint64_t* poly_1, uint64_t* poly_2, uint64_t* result, size_t poly_1_size, size_t poly_2_size){
        int i = threadIdx.x + blockIdx.x * blockDim.x;
        if (i >= poly_1_size + poly_2_size - 1) return;
        uint64_t sum = 0;
        for (int j = 0; j < poly_1_size; j++) {
            if (i - j >= 0 && i - j < poly_2_size) {
                sum += poly_1[j] * poly_2[i - j];
            }
        }
        result[i] = sum;
    }

}